#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include <stdio.h>
#include <iostream>

using namespace std;

__global__ void matrixMultiply(int *d_a, size_t pitch_a, int *d_b, size_t pitch_b, int *d_c, size_t pitch_c, const int N, const int M)
{
    __shared__ int input1Temp[4][3];
    __shared__ int input2Temp[3][4];

    int row = blockDim.y*blockIdx.y + threadIdx.y;
    int col = blockDim.x*blockIdx.x + threadIdx.x;

    if (row < N&&col < N)
    {
        // load d_a to shared memory
        if (col < N - 1)
        {
            int *shared_a = (int *)((char *)d_a + row*pitch_a) + col;
            input1Temp[row][col] = *shared_a;
            //__syncthreads();           
        }
        // load d_b to shared memory
        if (row < N - 1)
        {
            int *shared_b = (int *)((char *)d_b + row*pitch_b) + col;
            input2Temp[row][col] = *shared_b;
            __syncthreads();
        }

        int tmp = 0;
        for (size_t i = 0; i < M; i++)
        {
             tmp += input1Temp[row][i] * input2Temp[i][col];          
        }
        int *shared_c = (int *)((char *)d_c + row*pitch_c) + col;
        *shared_c = tmp;
    }

}


int main()
{
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    const int N = 4;
    const int M = 3;
    
    // use three streams to async copy array from host to device
    hipStream_t stream_a, stream_b, stream_c;
    hipStreamCreate(&stream_a); hipStreamCreate(&stream_b); hipStreamCreate(&stream_c);

    // allocate output array on device
    static int h_c[N][N];
    int *d_c;
    size_t pitch_c;
    hipMallocPitch(&d_c, &pitch_c, N * sizeof(int), N);
    hipMemcpy2DAsync(d_c, pitch_c, h_c, N * sizeof(int), N * sizeof(int), N, hipMemcpyHostToDevice, stream_c);

    // allocate 2d array on device
    int h_a[N][M] = { { 1,2,3 },{ 4,5,6 },{ 7,8,9 },{ 1,3,4 } };
    size_t pitch_a;
    int *d_a;
    hipMallocPitch(&d_a, &pitch_a, M * sizeof(int), N);
    hipMemcpy2DAsync(d_a, pitch_a, h_a, M * sizeof(int), M * sizeof(int), N, hipMemcpyHostToDevice, stream_a);

    int h_b[M][N] = { { 1,2,3,4 },{ 4,5,6,7 },{ 7,8,9,10 } };
    size_t pitch_b;
    int *d_b;
    hipMallocPitch(&d_b, &pitch_b, N * sizeof(int), M);
    hipMemcpy2DAsync(d_b, pitch_b, h_b, N * sizeof(int), N * sizeof(int), M, hipMemcpyHostToDevice, stream_b);

    hipStreamSynchronize(stream_a); hipStreamSynchronize(stream_b); hipStreamSynchronize(stream_c);

    dim3 blockSize(1);
    dim3 threadSize(N, N);
    matrixMultiply <<<blockSize, threadSize>>>(d_a, pitch_a, d_b, pitch_b, d_c, pitch_c, N, M);
    hipDeviceSynchronize();

    // copy result to host
    hipMemcpy2D(h_c, N * sizeof(int), d_c, pitch_c, N * sizeof(int), N, hipMemcpyDeviceToHost);

    for (size_t i = 0; i < N; i++)
    {
        for (size_t j = 0; j < N; j++)
        {
            cout << h_c[i][j] << ", ";
        }
        cout << endl;
    }

    //system("pause");
    hipFree(d_a); hipFree(d_b); hipFree(d_c);

    hipStreamDestroy(stream_a); hipStreamDestroy(stream_b); hipStreamDestroy(stream_c);
    return 0;
}