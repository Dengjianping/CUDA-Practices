
#include "hip/hip_runtime.h"


#include <iostream>
#include <chrono>
#include <stdio.h>

using namespace std;

const int N = 10000;

__global__ void addition(int *c, int *a, int *b, const int N)
{
    int i = blockDim.x*blockIdx.x + threadIdx.x;
    if (i < N )
    {
        c[i] = a[i] + b[i];
    }
}

int generateRandomNum(bool bigger)
{
    if (!bigger)
    {
        return rand() % 10; // range from 0 to 10
    }
    else 
    {
        return rand() % 100; // range from 0 to 10
    }
}

void initVec(int a[N], const int N, bool bigger=false)
{
    cout << "input array: ";
    for (int i = 0; i < N; i++)
    {
        a[i] = generateRandomNum(bigger);
        //cout << a[i] << ", ";
    }
    cout << endl << "------------" << endl;
}

void showArray(int *a, const int N)
{
    cout << "output array: ";
    for (size_t i = 0; i < N; i++)
    {
        cout << a[i] << ", ";
    }
    cout << endl << "------------" << endl;
}

void arrayAdditionOnCPU(int *c, int *a, int *b, const int N)
{
    for (size_t i = 0; i < N; i++)
    {
        c[i] = a[i] + b[i];
    }
}

int main()
{
    
    // host data
    int a[N], b[N], c[N], d[N];
    initVec(a, N);
    
    initVec(b, N, true);
    int size = N * sizeof(int);
    
    // device data
    int *d_a, *d_b, *d_c;
    // allocate space for devece data
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // copy data to device from host
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
    
    // define threads and blocks
    dim3 threadPerBlock(2); // just 1 thread in a block
    dim3 blockSize(N / threadPerBlock.x); // 2 blocks

    hipEvent_t start, stop;
    float deviceTimeCost;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    addition<<<blockSize, threadPerBlock>>>(d_c, d_a, d_b, N);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    
    hipEventElapsedTime(&deviceTimeCost, start, stop);
    //destroy all event
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
    // free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    cout << "Time Consumed on GPU: " << deviceTimeCost << endl;

    auto hostStart = chrono::steady_clock::now();
    arrayAdditionOnCPU(d, a, b, N);
    auto hostEnd = chrono::steady_clock::now();
    float hostTimeCost = chrono::duration_cast<chrono::duration<float> >(hostEnd - hostStart).count();
    cout << "Time Consumed on Host: " << hostTimeCost << endl;

    cout << "who is faster: " << deviceTimeCost / hostTimeCost << endl;

    //showArray(c, N);
    system("pause");

    return 0;
}