#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <iostream>
#include <cmath>
#include <vector>
#include <ctime>

using namespace std;
const int row = 2;
const int col = 2;

__global__ void addition(int *c,const int *a, const int *b)
{
	int i = blockDim.x*blockIdx.x + threadIdx.x;

	c[i] = a[i] + b[i];
}

int generateRandomNum(bool bigger)
{
	if (!bigger)
	{
		return rand() % 10;		
	}
	else 
	{
		return rand() % 100;
	}
}

void initVec(vector<int> & a, bool bigger==false)
{
	for (int i = 0; i < a.size(); i++)
	{
		a[i] = generateRandomNum(bigger);
	}
}

int main()
{
	// host data
	vector<int> a(10, 0);
	initVec(a);
	
	vector<int> b(10, 0);
	initVec(b, true);
	
	vector<int> c(10, 0);
	
	int size = a.size() * sizeof(int);
	
	// device data
	int *d_a, *d_b, *d_c;
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	// copy data to device from host
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	
	// define threads and blocks
	dim3 threadPerBlock(1);
	dim3 blockSize(10);

	addition<<<blockSize, threadPerBlock>>> (d_c, d_a, d_b);

	hipMemcpy(d_c, c, size, hipMemcpyDeviceToDevice);
	// memory free
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	cout << c[1][1] << c[1][0] << endl;
	system("pause");

	return 0;
}