
#include "hip/hip_runtime.h"


#include <iostream>
#include <cmath>
#include <vector>

using namespace std;
const int row = 2;
const int col = 2;

__global__ void addition(int c[row][col],const int a[row][col], const int b[row][col])
{
	int i = blockDim.y*blockIdx.y + threadIdx.y;
	int j = blockDim.x*blockIdx.x + threadIdx.x;

	c[i][j] = a[i][j] + b[i][j];
}

int main()
{
	// host data
	int a[row][col] = { { 1,2 },{ 3,4 } };
	int b[row][col] = { { 1,2 },{ 3,4 } };
	int c[row][col];
	int size = sizeof(a);

	// device data
	int d_a[row][col], d_b[row][col], d_c[row][col];
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	// copy data to device from host
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	
	// define threads and blocks
	dim3 threadPerBlock(1, 1);
	dim3 blockSize(2, 2);

	addition<<<blockSize, threadPerBlock>>> (d_c, d_a, d_b);

	hipMemcpy(d_c, c, size, hipMemcpyDeviceToDevice);
	// memory free
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	cout << c[1][1] << c[1][0] << endl;
	system("pause");

	return 0;
}