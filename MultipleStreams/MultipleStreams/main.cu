#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <thrust\device_vector.h>
#include <thrust\host_vector.h>

#include <stdio.h>
#include <iostream>

using namespace std;

const int N = 3;

__global__ void add(int *a, int N)
{
    int i = blockDim.x*blockIdx.x + threadIdx.x;
    if (i < N)
    {
        a[i] = a[i] + N;
    }
}

void showArray(int a[][3], const int N)
{
    cout << "show array: ";
    for (size_t i = 0; i < N; i++)
        for (size_t j = 0; j < N; j++)
        {
            cout << a[i][j] << ", ";
        }
    cout << endl;
}

void initStreams(thrust::host_vector<hipStream_t> & streams, const int N)
{
    for (size_t i = 0; i < N; i++)
    {
        hipStream_t stream;
        hipStreamCreate(&stream);
        streams.push_back(stream);
    }
}

void run(hipStream_t & stream, int *a, const int N)
{
    int *d_a;
    int size = N * sizeof(int);

    // blocks and threads
    dim3 blocks(N);
    dim3 threads(N);

    hipMalloc((void **)&d_a, size);
    hipStream_t copy;
    hipStreamCreateWithFlags(&copy, hipStreamNonBlocking);
    hipMemcpyAsync(d_a, a, size, hipMemcpyHostToDevice, copy); // asynchronizely copy data to device

    while (hipStreamQuery(copy) != hipSuccess)
    {}
    /*the same as using this api to hold here: 
    hipStreamSynchronize(copy);
    */
    
    add<<<blocks, threads, 0, stream>>> (d_a, N);
    hipStreamSynchronize(stream); // wait for stream done
    hipMemcpy(a, d_a, size, hipMemcpyDeviceToHost);
    hipFree(d_a);

    hipStreamDestroy(copy);
}

int main()
{
    thrust::host_vector<hipStream_t> streams;
    initStreams(streams, N);

    int a[N][N] = { {1,2,3},{4,5,6},{7,8,9} };

    for (size_t i = 0; i < streams.size(); i++)
    {
        run(streams[i], a[i], N);
    }

    showArray(a, N);
    // destroy all streams
    for (size_t i = 0; i < streams.size(); i++)
    {
        hipStreamDestroy(streams[i]);
    }

    system("pause");
    return 0;
}