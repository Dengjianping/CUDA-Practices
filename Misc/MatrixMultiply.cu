#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include <stdio.h>
#include <iostream>

using namespace std;

class Matrix
{
private:
    int row, col;
public:
    thrust::host_vector<thrust::host_vector<int> > v;
    Matrix();
    Matrix(const int r, const int c);
    int randNum() { return rand() % 100; };
    int rows() { return row; };
    int cols() { return col; };
    void show() const;
    ~Matrix();
};

Matrix::Matrix()
{
    row = col = 0;
    for (size_t i = 0; i < row; i++)
    {
        thrust::host_vector<int> t;
        for (size_t j = 0; j < col; j++)
        {
            t.push_back(this->randNum());
        }
        v.push_back(t);
    }
}

Matrix::Matrix(const int r, const int c)
{
    row = r, col = c;
    for (size_t i = 0; i < row; i++)
    {
        thrust::host_vector<int> t;
        for (size_t j = 0; j < col; j++)
        {
            t.push_back(this->randNum());
        }
        v.push_back(t);
    }
}

void Matrix::show() const
{
    for (size_t i = 0; i < row; i++)
    {
        for (size_t j = 0; j < col; j++)
        {
            cout << v[i][j] << ", ";
        }
        cout << endl;
    }
}

Matrix::~Matrix()
{
}

__global__ void matrixMultiply(int *d_a, size_t pitch_a, int *d_b, size_t pitch_b, int *d_c, size_t pitch_c, const int N, const int M)
{
    __shared__ int input1Temp[4][3];
    __shared__ int input2Temp[3][4];

    int row = blockDim.y*blockIdx.y + threadIdx.y;
    int col = blockDim.x*blockIdx.x + threadIdx.x;

    if (row < N&&col < N)
    {
        // load d_a to shared memory
        if (col < N - 1)
        {
            int *shared_a = (int *)((char *)d_a + row*pitch_a) + col;
            input1Temp[row][col] = *shared_a;
            __syncthreads();
            printf("input1Temp[%d][%d]: %d\n", row, col, input1Temp[row][col]);
        }
        // load d_b to shared memory
        if (row < N - 1)
        {
            int *shared_b = (int *)((char *)d_b + row*pitch_b) + col;
            input2Temp[row][col] = *shared_b;
            __syncthreads();
            printf("input2Temp[%d][%d]: %d\n", row, col, input2Temp[row][col]);
        }

        int tmp = 0;
        for (size_t i = 0; i < N; i++)
        {
            tmp += input1Temp[row][i] * input2Temp[i][col];
        }
        //shared_c[col*pitch_c + row] = tmp;
        //d_c[row*pitch_c+col] = tmp;
        int *shared_c = (int *)((char *)d_c + row*pitch_c) + col;
        *shared_c = tmp;
    }

}

__global__ void showPitch(int *a, size_t pitch, int rows, int cols)
{
    int row = blockDim.y*blockIdx.y + threadIdx.y;
    int col = blockDim.x*blockIdx.x + threadIdx.x;

    if (row < rows&&col < cols)
    {
        int *t = (int *)((char *)a + row*pitch) + col;
        printf("a[%d][%d]: %d", row, col, *t);
    }
}

int main()
{
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    const int N = 4;
    const int M = 3;
    
    // use three streams to async copy array from host to device
    hipStream_t stream_a, stream_b, stream_c;
    hipStreamCreate(&stream_a); hipStreamCreate(&stream_b); hipStreamCreate(&stream_c);

    // allocate output array on device
    static int h_c[N][N];
    int *d_c;
    size_t pitch_c;
    hipMallocPitch(&d_c, &pitch_c, N * sizeof(int), N);
    hipMemcpy2DAsync(d_c, pitch_c, h_c, N * sizeof(int), N * sizeof(int), N, hipMemcpyHostToDevice, stream_c);

    // allocate 2d array on device
    int h_a[N][M] = { { 1,2,3 },{ 4,5,6 },{ 7,8,9 },{ 1,3,4 } };
    size_t pitch_a;
    int *d_a;
    hipMallocPitch(&d_a, &pitch_a, M * sizeof(int), N);
    hipMemcpy2DAsync(d_a, pitch_a, h_a, M * sizeof(int), M * sizeof(int), N, hipMemcpyHostToDevice, stream_a);

    int h_b[M][N] = { { 1,2,3,4 },{ 4,5,6,7 },{ 7,8,9,10 } };
    size_t pitch_b;
    int *d_b;
    hipMallocPitch(&d_b, &pitch_b, N * sizeof(int), M);
    hipMemcpy2DAsync(d_b, pitch_b, h_b, N * sizeof(int), N * sizeof(int), M, hipMemcpyHostToDevice, stream_b);

    hipStreamSynchronize(stream_a); hipStreamSynchronize(stream_b); hipStreamSynchronize(stream_c);

    cout << "------------------" << endl;
    dim3 blockSize(1);
    dim3 threadSize(N, N);
    cout << threadSize.x << endl;
    //showPitch <<<blockSize, threadSize>>> (d_a, pitch, N, M);
    matrixMultiply <<<blockSize, threadSize>>>(d_a, pitch_a, d_b, pitch_b, d_c, pitch_c, N, M);
    hipDeviceSynchronize();

    // copy result to host
    hipMemcpy2D(h_c, M * sizeof(int), d_c, pitch_c, M * sizeof(int), N, hipMemcpyDeviceToHost);

    for (size_t i = 0; i < N; i++)
    {
        for (size_t j = 0; j < M; j++)
        {
            cout << h_c[i][j] << ", ";
        }
        cout << endl;
    }

    system("pause");
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    hipStreamDestroy(stream_a); hipStreamDestroy(stream_b); hipStreamDestroy(stream_c);
    return 0;
}