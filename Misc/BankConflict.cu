#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>

using namespace std;

const int N = 1024;
//__managed__ int a[N];

class TimeRecord
{
private:
    hipEvent_t start, end;
    float time;
public:
    TimeRecord();
    void startRecord();
    void endRecord();
    float timeCost();
    ~TimeRecord();
};

TimeRecord::TimeRecord()
{
    hipEventCreate(&start);
    hipEventCreate(&end);
    time = 0.0;
}

void TimeRecord::startRecord()
{
    hipEventRecord(this->start);
}

void TimeRecord::endRecord()
{
    hipEventRecord(this->end);
    hipEventSynchronize(this->end); //wait end event to finish
}

float TimeRecord::timeCost()
{
    hipEventElapsedTime(&this->time, start, end);
    return this->time;
}

TimeRecord::~TimeRecord()
{
    hipEventDestroy(start);
    hipEventDestroy(end);
}


__constant__ int n = 10;

__device__ int returnValue()
{
    return 10;
}

__global__ void bankConflict(int *a, const int warp, const int N)
{
    __shared__ int sharedData[32][32];

    int row = blockDim.y*blockIdx.y + threadIdx.y;
    int col = blockDim.x*blockIdx.x + threadIdx.x;

    int index = row*warp + col;
    if (index > 1022)
    {
        printf("a[index]: %d, %d, %d\n", index, row,col);

    }

    sharedData[row][col] = a[index];
    __syncthreads();
    if (row < warp&&col < warp)
    {
        a[index] = sharedData[col][row] + n;
        //printf("a[index]: %d\n", a[index]);
    }
}

__global__ void nonBankConflict(int *a, const int warp, const int N)
{
    __shared__ int sharedData[32][32];

    int row = blockDim.y*blockIdx.y + threadIdx.y;
    int col = blockDim.x*blockIdx.x + threadIdx.x;

    int index = row*warp + col;

    sharedData[row][col] = a[index];
    __syncthreads();
    if (row < warp&&col < warp)
    {
        a[index] = sharedData[row][col] + n;
        printf("a[index]: %d\n", a[index]);
    }
}

void initArray(int *a, int N)
{
    for (size_t i = 0; i < N; i++)
    {
        a[i] = rand() % 100;
    }
}

void zeroCopy(hipDeviceProp_t *prop, int *hostData, const int N)
{
    if (!prop->canMapHostMemory)
    {
        cout << "your device cannot support map host memory to device" << endl;
        return;
    }
    hipHostAlloc(&hostData, N * sizeof(int), hipHostMallocMapped);
    initArray(hostData, N);
}


int main()
{
    int count;
    hipDeviceProp_t prop;

    hipGetDeviceCount(&count);
    hipGetDeviceProperties(&prop, 0);

    int warp = prop.warpSize;
    dim3 blockSize(1);
    dim3 threadSize(warp, warp);

    int *hostData;
    //zeroCopy(&prop, hostData, N);
    if (!prop.canMapHostMemory)
    {
        cout << "your device cannot support map host memory to device" << endl;
        return;
    }
    hipHostAlloc(&hostData, N * sizeof(int), hipHostMallocMapped);
    initArray(hostData, N);

    cout << hostData[0] << ", " << hostData[1] << endl;

    //TimeRecord noConflict;
    //noConflict.startRecord();
    hipEvent_t start, stop;
    float deviceTimeCost;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    nonBankConflict << <blockSize, threadSize >> >(hostData, warp, N);
    //bankConflict <<<blockSize,threadSize>>>(hostData, warp, N);
    //noConflict.endRecord();
    hipEventRecord(stop);

    hipEventSynchronize(stop);

    hipEventElapsedTime(&deviceTimeCost, start, stop); // friendly warning here returns in millisecond
                                                        //destroy all event
    hipEventDestroy(start);
    hipEventDestroy(stop);

    cout << "time cost with bank conflict: " << deviceTimeCost << endl;
    cout << hostData[0] << ", " << hostData[1] << endl;


    hipHostFree(hostData);

    system("pause");
    return 0;
}