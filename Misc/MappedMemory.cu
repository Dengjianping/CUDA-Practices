#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <thrust\device_vector.h>
#include <thrust\host_vector.h>

#include <stdio.h>
#include <iostream>

using namespace std;

__global__ void kernel(int *, const int N)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < N)
	{
		a[i] = a[i] + 3;
	}
}

void showArray(int *a, const int N)
{
	for (int i = 0; i < N; i++)
	{
		cout << a[i] << ", ";
	}
	cout << endl;
}

void run()
{
	const int N = 20;
	static int h_a[N];
	int *d_a;
	int size = N * sizeof(int);
	
	dim3 blocks(10);
	dim3 threads(2);
	
	int count;
	cudaDeviceCount(&count);
	cudaDeviceProperties prop;
	if (count == 1)
	{
		hipGetDeviceProperties(&prop, count - 1);
	}
	
	if (!prop.canMapHostMemory)
	{
		// cudaHostMalloc(&h_a, size, hipHostMallocMapped);
		printf("cannot use map memory")
		return;
	}
	cudaHostMalloc(&h_a, size, hipHostMallocMapped);
	hipHostGetDevicePointer(&d_a, h_a, 0);
	kernel<<<blocks, threads>>>(d_a, N);
	
	hipDeviceSynchronize();
	hipHostFree(h_a);
	
	cout << "result: " << endl;
	showArray(h_a, N);
	
	system("pause");
	return 0;
}