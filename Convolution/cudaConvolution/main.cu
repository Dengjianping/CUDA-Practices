#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "thrust/device_vector.h"
#include "thrust/host_vector.h"

#include <stdio.h>
#include <iostream>

#define hostMatrix thrust::host_vector<thrust::host_vector<int> >
#define deviceMatrix thrust::device_vector<thrust::device_vector<int> >

using namespace std;

__global__ void convolution1D(int *d_input1D, const int P, int *d_kernel1D, const int M, int *d_output1D)
{
    int row = blockDim.y*blockIdx.y + threadIdx.y;
    int col = blockDim.x*blockIdx.x + threadIdx.x;

    int index = row*blockDim.x + col;
    extern __shared__ int sharedInput1D[];
    if (index < P)
    {
        sharedInput1D[index] = d_input1D[index];
    }

    extern __shared__ int sharedKernel1D[];
    if (index < M)
    {
        sharedKernel1D[index+P] = d_kernel1D[index];
        __syncthreads();
    }

    if (index < P)
    {
        for (size_t i = 0; i < M; i++)
        {
            d_output1D[index + i] += sharedKernel1D[i+P] * sharedInput1D[index];
            // about how to retrieve multiple share memory in 
        }
    }
}

__global__ void convolution2D(int *d_input2D, size_t inputPitch, const int P, const int Q, int *d_kernel2D, size_t kernelPitch, const int M, const int N, int *d_output2D, size_t outputPitch)
{
    __shared__ int kernel[3][4];
    __shared__ int input[4][4];

    int row = blockDim.y*blockIdx.y + threadIdx.y;
    int col = blockDim.x*blockIdx.x + threadIdx.x;

    if (row < P&&col < Q)
    {
        // load kernel data to share memory
        if (row < M&&col < N)
        {
            int *sharedKernel = (int *)((char *)d_kernel2D +row*kernelPitch) + col;
            kernel[row][col] = *sharedKernel;
            __syncthreads();
        }

        // load input data to share memory
        int *sharedInput = (int *)((char *)d_input2D +row*inputPitch) + col;
        input[row][col] = *sharedInput;
        __syncthreads();


        // convolving
        int r = 0, c = 0;
        for (size_t i = 0; i < M; i++)
        {
            for (size_t j = 0; j < N; j++)
            {
                int *sharedOutput = (int *)((char *)d_output2D + (row + i)*outputPitch) + (col + j);
                *sharedOutput += kernel[i][j] * input[row][col];
            }
        }
    }
}

void initArray(int *a, const int N)
{
    for (size_t i = 0; i < N; i++)
    {
        a[i] = rand() % 10;
    }
}

void showArray(int *a, const int N)
{
    for (size_t i = 0; i < N; i++)
    {
        cout << a[i] << ", ";
    }
    cout << endl;
}

void run()
{
    hipStream_t kernlStream, inputStream, outputStream;
    hipStreamCreate(&kernlStream); hipStreamCreate(&inputStream); hipStreamCreate(&outputStream);
    // copy kernel data to device
    const int M = 3, N = 4;
    //int kernel2D[M][N] = { { 1,2,3,4 },{ 3,4,5,6 },{ 6,7,8,9 } };
    int kernel2D[M][N] = { {1,2,3,4}, {3,4,5,6},{6,7,8,9} };
    int *d_kernel2D;
    size_t kernelPitch;
    hipMallocPitch(&d_kernel2D, &kernelPitch, N * sizeof(int), M);
    hipMemcpy2DAsync(d_kernel2D, kernelPitch, kernel2D, N * sizeof(int), N * sizeof(int), M, hipMemcpyHostToDevice, kernlStream);

    // copy input data to device
    const int P = 4, Q = 4;
    int input2D[P][Q] = { { 1,2,3,4 },{ 3,4,5,6 },{ 6,7,8,9 },{10,3,7,6 } };
    int *d_input2D;
    size_t inputPitch;
    hipMallocPitch(&d_input2D, &inputPitch, Q * sizeof(int), P);
    // about how to calculate source pitch size, see offcial document
    hipMemcpy2DAsync(d_input2D, inputPitch, input2D, Q * sizeof(int), Q * sizeof(int), P, hipMemcpyHostToDevice, inputStream);

    // init output data
    static int output2D[P + M - 1][Q + N - 1] = { {} };
    int *d_output2D;
    size_t outputPitch;
    hipMallocPitch(&d_output2D, &outputPitch, (Q + N - 1) * sizeof(int), P + M - 1);
    hipMemcpy2DAsync(d_output2D, outputPitch, output2D, (Q + N - 1) * sizeof(int), (Q + N - 1) * sizeof(int), P + M - 1, hipMemcpyHostToDevice, outputStream);

    // hold here to wait for all data is tansfered completely
    hipStreamSynchronize(kernlStream); hipStreamSynchronize(inputStream); hipStreamSynchronize(outputStream);

    // define block size and thread size
    dim3 blockSize(1);
    dim3 threadSize(8, 8);
    convolution2D<<<blockSize, threadSize>>>(d_input2D, inputPitch, P, Q, d_kernel2D, kernelPitch, M, N, d_output2D, outputPitch);
    hipError_t error = hipDeviceSynchronize();
    if (error != hipSuccess)
    {
        cout << hipGetErrorString(error) << endl;
    }
    
    // hold host execution until device compution finish
    error = hipMemcpy2D(output2D, (Q + N - 1) * sizeof(int), d_output2D, outputPitch, (Q + N - 1) * sizeof(int), P + M - 1, hipMemcpyDeviceToHost);
    if (error != hipSuccess)
    {
        cout << hipGetErrorString(error) << endl;
    }
    
    // clean up
    hipFree(d_input2D); hipFree(d_kernel2D); hipFree(d_output2D);
    hipStreamDestroy(kernlStream); hipStreamDestroy(inputStream); hipStreamDestroy(outputStream);

    for (size_t i = 0; i < P+M-1; i++)
    {
        for (size_t j = 0; j < Q+N-1; j++)
        {
            cout << output2D[i][j] << ", ";
        }
        cout << endl;
    }
}

int main()
{
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    int warp = prop.warpSize;
    // for 1D convolution
    cout << "1D array convolution: " << endl;
    const int M = 4, N = 3;
    static int *kernel1D;
    int *d_kernel1D;
    hipHostAlloc(&kernel1D, M * sizeof(int), hipHostMallocMapped); // make sure your device support host memory map device
    initArray(kernel1D, M);
    cout << "kernel array: " << endl;
    showArray(kernel1D, M);
    hipHostGetDevicePointer(&d_kernel1D, kernel1D, 0);

    const int P = 10, Q = 10;
    static int *intput1D;
    int *d_input1D;
    hipHostAlloc(&intput1D, P * sizeof(int), hipHostMallocMapped);
    initArray(intput1D, P);
    cout << endl << "input array: " << endl;
    showArray(intput1D, P);
    hipHostGetDevicePointer(&d_input1D, intput1D, 0);

    static int *output1D;
    int *d_output1D;
    hipHostAlloc(&output1D, (P + M - 1) * sizeof(int), hipHostMallocMapped);
    hipHostGetDevicePointer(&d_output1D, output1D, 0);

    int size = (P + M - 1) % 2 == 0 ? P + M - 1 : P + M;
    dim3 block1D(1);
    // actually you better choose a 32 multiple of number
    dim3 thread1D(2, size / 2);
    int dynamicShareMemSize = (P + M) * sizeof(int);
    convolution1D<<<block1D,thread1D, dynamicShareMemSize >>>(d_input1D, P, d_kernel1D, M, d_output1D);

    hipDeviceSynchronize();
    cout << endl << "output array: " << endl;
    showArray(output1D, P + M - 1);

    hipHostFree(d_kernel1D); hipHostFree(d_input1D); hipHostFree(d_output1D);

    // for 2-dim convolution
    cout << "--------------------" << endl;
    cout << "2D array convolution: " << endl;
    
    run();

    system("pause");
    return 0;
}